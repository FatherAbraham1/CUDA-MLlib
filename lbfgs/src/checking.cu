/*
 * Licensed to the Apache Software Foundation (ASF) under one or more
 * contributor license agreements.  See the NOTICE file distributed with
 * this work for additional information regarding copyright ownership.
 * The ASF licenses this file to You under the Apache License, Version 2.0
 * (the "License"); you may not use this file except in compliance with
 * the License.  You may obtain a copy of the License at
 *
 *    http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../include/cuda_checking.h"
#include <stdlib.h>
#include <stdio.h>
#include <sstream>
#include <iostream>



  // This a copy of _cudaGetErrorEnum() from cuda-7.0/samples/common/inc/helpers_cuda.h
  const char *cublasGetErrorString(hipblasStatus_t e)
  {
    switch (e) {
    case HIPBLAS_STATUS_SUCCESS:          return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:  return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:     return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:    return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:    return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:    return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:   return "HIPBLAS_STATUS_INTERNAL_ERROR";
    default: break;
    }
    
    static char answer[40];
    sprintf(answer, "<Unknown hipblasStatus_t %d>", e);
    return answer;
  }

                 
