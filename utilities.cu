#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one or more
 * contributor license agreements.  See the NOTICE file distributed with
 * this work for additional information regarding copyright ownership.
 * The ASF licenses this file to You under the Apache License, Version 2.0
 * (the "License"); you may not use this file except in compliance with
 * the License.  You may obtain a copy of the License at
 *
 *    http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include "utilities.h"

struct GPUState {
	int deviceCount; // number of GPUs to use
	int deviceToUse; // GPU to use (round-robin)
	pthread_mutex_t initLock;
} gpustate = {-1,-1,PTHREAD_MUTEX_INITIALIZER};

// like hipMalloc, but tried GPU, and if fails, mallocs on the host instead and registers
// memory allocated via this method should be freed with freeBest
hipError_t mallocBest ( void **devPtr, size_t size ) {
	hipError_t returnVal = hipSuccess;
	if (hipMalloc(devPtr, size) != hipSuccess) {
		//fprintf(stderr, "Unable to malloc %i bytes on the device - using host memory\n", size);
		void* h_ptr;
		returnVal = hipHostAlloc(&h_ptr, size, hipHostMallocMapped | hipHostMallocWriteCombined);
		if (returnVal == hipSuccess) checkCudaErrors(hipHostGetDevicePointer(devPtr, h_ptr, 0));
	}
	return returnVal;
}

// like hipFree, but can also handle host pointers.  To be used with mallocBest
hipError_t freeBest ( void *devPtr ) {
	hipPointerAttribute_t attributes;
	checkCudaErrors(hipPointerGetAttributes (&attributes, devPtr));
	//fprintf(stderr, "freeing memory of type %i\n", attributes.memoryType);
	if (attributes.memoryType == hipMemoryTypeDevice)
		checkCudaErrors(hipFree(devPtr));
	else
		checkCudaErrors(hipHostFree(devPtr));

	return hipSuccess;
}

// returns which GPU to run on, or -1 if no GPUs are available
int get_gpu() {
	if (gpustate.deviceCount == 1)
		return 0; // return immediately for the common case of 1 GPU
	else if (gpustate.deviceCount > 1) { // multiple GPUs
		int newval, oldval;
		do {
			oldval = gpustate.deviceToUse;
			if (oldval == gpustate.deviceCount-1)
				newval = 0;
			else
				newval = oldval+1;
		} while (!__sync_bool_compare_and_swap(&gpustate.deviceToUse, oldval, newval));
	}
	else if (gpustate.deviceCount == -1) { // not yet initialized... run initialization
		pthread_mutex_lock(&gpustate.initLock);
		// check if another thread already completed initialization
		if (gpustate.deviceCount != -1) {
			pthread_mutex_unlock(&gpustate.initLock);
			return get_gpu();
		}
		// continue with initialization
		if (hipGetDeviceCount(&gpustate.deviceCount)) {
			fprintf(stderr, "Cuda Error in GetDeviceCount: %s\n", hipGetErrorString(hipGetLastError()));
			gpustate.deviceCount = 0;
		}
		else if (gpustate.deviceCount <= 0)
			gpustate.deviceCount = 0;
		else
			gpustate.deviceToUse = 0;

		for (int deviceID=0; deviceID<gpustate.deviceCount; deviceID++) {
			hipSetDevice(deviceID);
			hipDeviceReset();
		}
		pthread_mutex_unlock(&gpustate.initLock);
	}

	return gpustate.deviceToUse;
}
